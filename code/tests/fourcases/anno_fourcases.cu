#include "hip/hip_runtime.h"
/* Norman Ponte; Joey Fernau
 * annotation generation test
 */

#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include <getopt.h>
#include <string>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include "../../lib/CycleTimer.h"

<<<<<<< HEAD
extern float toBW(int bytes, float sec);

__device__ int test ( int x , int y , int z ) {
  printf("%d,%d\n", blockIdx.x * blockDim.x + threadIdx.x, 2);
  int result = 0;
  if (x == 0) {
    printf("%d,%d\n", blockIdx.x * blockDim.x + threadIdx.x, 3);
    for (int i = 0; i < 10000; i++)
      result += y - z;
  } else if (x == 1) {
    printf("%d,%d\n", blockIdx.x * blockDim.x + threadIdx.x, 4);
    for (int i = 0; i < 10000; i++)
      result += y + z;
  } else if (x == 2) {
    printf("%d,%d\n", blockIdx.x * blockDim.x + threadIdx.x, 5);
    for (int i = 0; i < 10000; i++)
      result += y * z;
  } else {
    printf("%d,%d\n", blockIdx.x * blockDim.x + threadIdx.x, 6);
    for (int i = 0; i < 10000; i++)
      result += y / z;
  }
  printf("%d,%d\n", blockIdx.x * blockDim.x + threadIdx.x, 7);
=======
#define BBLOG(bbid) printf("%d,%d\n", blockIdx.x * blockDim.x + threadIdx.x, bbid)

extern float toBW(int bytes, float sec);

__device__ int test ( int x , int y , int z ) {
  BBLOG(2);
  int result = 0;
  if (x == 0) {
    BBLOG(3);
    for (int i = 0; i < 1000000; i++)
      result += y - z;
  } else if (x == 1) {
    BBLOG(4);
    for (int i = 0; i < 1000000; i++)
      result += y + z;
  } else if (x == 2) {
    BBLOG(5);
    for (int i = 0; i < 1000000; i++)
      result += y * z;
  } else {
    BBLOG(6);
    for (int i = 0; i < 1000000; i++)
      result += y / z;
  }
  BBLOG(7);
>>>>>>> 046f3e2357ae5d625cbe41cd94ef940ec1ed40c4
  return result;
}

__global__ void
test_kernel(int N, float* result) {
<<<<<<< HEAD
    printf("%d,%d\n", blockIdx.x * blockDim.x + threadIdx.x, 0);
=======
    BBLOG(0);
>>>>>>> 046f3e2357ae5d625cbe41cd94ef940ec1ed40c4
    // compute overall index from position of thread in current block,
    // and given the block we are in
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < N) {
<<<<<<< HEAD
       printf("%d,%d\n", blockIdx.x * blockDim.x + threadIdx.x, 1);
       result[index] = test(index % 4, index % 13, index % 7);
    }
    printf("%d,%d\n", blockIdx.x * blockDim.x + threadIdx.x, 8);
=======
        BBLOG(1);
       result[index] = test(index % 4, index % 13, index % 7);
    }
    BBLOG(8);
>>>>>>> 046f3e2357ae5d625cbe41cd94ef940ec1ed40c4
}

void
mainCuda(int N, float* resultarray) {

<<<<<<< HEAD
    int totalBytes = sizeof(float) * N;

=======
>>>>>>> 046f3e2357ae5d625cbe41cd94ef940ec1ed40c4
    // compute number of blocks and threads per block
    const int threadsPerBlock = 32;
    const int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;

    float* device_result;
<<<<<<< HEAD
    hipMalloc((void **) &device_result, totalBytes);

    // start timing after allocation of device memory.
    //double startTime = CycleTimer::currentSeconds();
=======
    hipMalloc((void **) &device_result, N * sizeof(float));

    // start timing after allocation of device memory.
    double startTime = CycleTimer::currentSeconds();
>>>>>>> 046f3e2357ae5d625cbe41cd94ef940ec1ed40c4

    //hipMemcpy(device_x, xarray, N * sizeof(float),
    //           hipMemcpyHostToDevice);

<<<<<<< HEAD
    //double kernelStartTime = CycleTimer::currentSeconds();
    test_kernel<<<blocks, threadsPerBlock>>>(N, device_result);
    hipDeviceSynchronize();
    //double kernelEndTime = CycleTimer::currentSeconds();
=======

    test_kernel<<<blocks, threadsPerBlock>>>(N, device_result);
    hipDeviceSynchronize();
>>>>>>> 046f3e2357ae5d625cbe41cd94ef940ec1ed40c4

    hipMemcpy(resultarray, device_result, N * sizeof(float),
               hipMemcpyDeviceToHost);

<<<<<<< HEAD
    double endTime = CycleTimer::currentSeconds();

=======
>>>>>>> 046f3e2357ae5d625cbe41cd94ef940ec1ed40c4
    hipError_t errCode = hipPeekAtLastError();
    if (errCode != hipSuccess) {
        fprintf(stderr, "WARNING: A CUDA error occured: code=%d, %s\n",
                errCode, hipGetErrorString(errCode));
    }
<<<<<<< HEAD
    //double kernelDuration = kernelEndTime - kernelStartTime;
    //printf("Kernel time: %.3f ms\t\t[%.3f GB/s]\n", 1000.f * kernelDuration, toBW(totalBytes, kernelDuration));
    //double overallDuration = endTime - startTime;
    //printf("Overall time: %.3f ms\t\t[%.3f GB/s]\n", 1000.f * overallDuration, toBW(totalBytes, overallDuration));

    /*
    std::cout << "{ ";
    for (int i = 0; i < N; i++) {
        std::cout << resultarray[i] << ", ";
    } std::cout << " }" << std::endl;
    */
=======
>>>>>>> 046f3e2357ae5d625cbe41cd94ef940ec1ed40c4

    hipFree(device_result);
}

// return GB/s
float toBW(int bytes, float sec) {
   return static_cast<float>(bytes) / (1024. * 1024. * 1024.) / sec;
}

void mainCuda(int N, float* result);

int main(int argc, char** argv)
{
<<<<<<< HEAD

    int N = 1024;

    float* resultarray = new float[N];

    printf("tid,bb\n");
=======
    printf("tid,bb\n");
    int N = std::atoi(argv[1]);

    float* resultarray = new float[N];

>>>>>>> 046f3e2357ae5d625cbe41cd94ef940ec1ed40c4
    mainCuda(N, resultarray);

    return 0;
}


