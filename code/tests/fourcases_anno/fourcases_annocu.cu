#include "hip/hip_runtime.h"
/* Norman Ponte; Joey Fernau
 * annotation generation test
 */

#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include <getopt.h>
#include <string>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include "../../lib/CycleTimer.h"

extern float toBW(int bytes, float sec);

__device__ int test ( int x , int y , int z ) {
  printf("%d %d\n", blockIdx.x * blockDim.x + threadIdx.x, 2);
  int result = 0;
  if (x == 0) {
    printf("%d %d\n", blockIdx.x * blockDim.x + threadIdx.x, 3);
    for (int i = 0; i < 10000; i++)
      result += y - z;
  } else if (x == 1) {
    printf("%d %d\n", blockIdx.x * blockDim.x + threadIdx.x, 4);
    for (int i = 0; i < 10000; i++)
      result += y + z;
  } else if (x == 2) {
    printf("%d %d\n", blockIdx.x * blockDim.x + threadIdx.x, 5);
    for (int i = 0; i < 10000; i++)
      result += y * z;
  } else {
    printf("%d %d\n", blockIdx.x * blockDim.x + threadIdx.x, 6);
    for (int i = 0; i < 10000; i++)
      result += y / z;
  }
  printf("%d %d\n", blockIdx.x * blockDim.x + threadIdx.x, 7);
  return result;
}

__global__ void
test_kernel(int N, float* result) {
    printf("%d %d\n", blockIdx.x * blockDim.x + threadIdx.x, 0);
    // compute overall index from position of thread in current block,
    // and given the block we are in
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < N) {
       printf("%d %d\n", blockIdx.x * blockDim.x + threadIdx.x, 1);
       result[index] = test(index % 4, index % 13, index % 7);
    }
    printf("%d %d\n", blockIdx.x * blockDim.x + threadIdx.x, 8);
}

void
mainCuda(int N, float* resultarray) {

    int totalBytes = sizeof(float) * N;

    // compute number of blocks and threads per block
    const int threadsPerBlock = 32;
    const int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;

    float* device_result;
    hipMalloc((void **) &device_result, N * sizeof(float));

    // start timing after allocation of device memory.
    double startTime = CycleTimer::currentSeconds();

    //hipMemcpy(device_x, xarray, N * sizeof(float),
    //           hipMemcpyHostToDevice);

    double kernelStartTime = CycleTimer::currentSeconds();
    test_kernel<<<blocks, threadsPerBlock>>>(N, device_result);
    hipDeviceSynchronize();
    double kernelEndTime = CycleTimer::currentSeconds();

    hipMemcpy(resultarray, device_result, N * sizeof(float),
               hipMemcpyDeviceToHost);

    double endTime = CycleTimer::currentSeconds();

    hipError_t errCode = hipPeekAtLastError();
    if (errCode != hipSuccess) {
        fprintf(stderr, "WARNING: A CUDA error occured: code=%d, %s\n",
                errCode, hipGetErrorString(errCode));
    }
    double kernelDuration = kernelEndTime - kernelStartTime;
    printf("Kernel time: %.3f ms\t\t[%.3f GB/s]\n", 1000.f * kernelDuration, toBW(totalBytes, kernelDuration));
    double overallDuration = endTime - startTime;
    printf("Overall time: %.3f ms\t\t[%.3f GB/s]\n", 1000.f * overallDuration, toBW(totalBytes, overallDuration));

    std::cout << "{ ";
    for (int i = 0; i < N; i++) {
        std::cout << resultarray[i] << ", ";
    } std::cout << " }" << std::endl;

    hipFree(device_result);
}

// return GB/s
float toBW(int bytes, float sec) {
   return static_cast<float>(bytes) / (1024. * 1024. * 1024.) / sec;
}

void mainCuda(int N, float* result);

int main(int argc, char** argv)
{

    int N = 64;

    float* resultarray = new float[N];

    mainCuda(N, resultarray);

    return 0;
}


